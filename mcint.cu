//# define THRUST_DEVICE_SYSTEM THRUST_DEVICE_SYSTEM_OMP

#include <thrust/functional.h> // function objects & tools
#include <thrust/random.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include "hip/hip_runtime_api.h"

/**********************************************************/

#ifdef _WIN32

#define WINDOWS_LEAN_AND_MEAN
#include <windows.h>

typedef LARGE_INTEGER app_timer_t;

static inline void timer(app_timer_t *t_ptr)
{
#ifdef __HIPCC__
  checkCudaErrors(hipDeviceSynchronize());
#endif
  QueryPerformanceCounter(t_ptr);
}

double elapsed_time(app_timer_t start, app_timer_t stop)
{
  LARGE_INTEGER clk_freq;
  QueryPerformanceFrequency(&clk_freq);
  return (stop.QuadPart - start.QuadPart) /
         (double) clk_freq.QuadPart * 1e3;
}

#else

#include <time.h> /* requires linking with rt library
                     (command line option -lrt) */

typedef struct timespec app_timer_t;

static inline void timer(app_timer_t *t_ptr) {
#ifdef __HIPCC__
    checkCudaErrors(hipDeviceSynchronize());
#endif
    clock_gettime(CLOCK_MONOTONIC, t_ptr);
}

double elapsed_time(app_timer_t start, app_timer_t stop) {
    return 1e+3 * (stop.tv_sec - start.tv_sec) +
           1e-6 * (stop.tv_nsec - start.tv_nsec);
}

#endif

/**********************************************************/

class randuni :
        public thrust::unary_function<unsigned long long, float> {
private:
    thrust::default_random_engine rng;
    thrust::uniform_real_distribution<float> uni;
public:
    randuni(unsigned int seed, float a = 0.0f, float b = 1.0f) :
            rng(seed), uni(a, b) {}

    __host__ __device__

    float operator()(unsigned long long i) {
        rng.discard(i); // odrzuæ liczby z "poprzednich" w¹tków
        return uni(rng);
    }
};

/**********************************************************/

typedef thrust::tuple<float, float, float> point3D;

struct fun : public thrust::unary_function<point3D, float> {
    __host__ __device__

    float operator()(const point3D &p) const {
        float x = thrust::get<0>(p);
        float y = thrust::get<1>(p);
        float z = thrust::get<2>(p);
        float s = x * x + y * y + z * z;
        if (s <= 1) return 1;
        else return 0;
    }
};

/**********************************************************/

int main() {
    app_timer_t t0, t1, t2, t3;
    float integral;
    timer(&t0); //--------------------------------------------
    thrust::device_vector<float> x(1000), y(x.size()), z(x.size());
    timer(&t1); //--------------------------------------------
    randuni gen_x(40, -1.0f, 1.0f);
    randuni gen_y(41, -1.0f, 1.0f);
    randuni gen_z(42, -1.0f, 1.0f);

    thrust::transform(thrust::make_counting_iterator<unsigned long long>(0),
                      thrust::make_counting_iterator<unsigned long long>(x.size()),
                      x.begin(), gen_x);

    thrust::transform(thrust::make_counting_iterator<unsigned long long>(0),
                      thrust::make_counting_iterator<unsigned long long>(y.size()),
                      y.begin(), gen_y);

    thrust::transform(thrust::make_counting_iterator<unsigned long long>(0),
                      thrust::make_counting_iterator<unsigned long long>(z.size()),
                      z.begin(), gen_z);

    timer(&t2); //--------------------------------------------

    integral = thrust::transform_reduce(
            thrust::make_zip_iterator(thrust::make_tuple(x.begin(), y.begin(), z.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(x.end(), y.end(), z.end())),
            fun(),
            0.0f,
            thrust::plus<float>()
    ) * 8 / 1000;

    timer(&t3); //--------------------------------------------

    using std::cout;
    using std::endl;
    cout << "pi = " << 0.75f * integral << std::endl; // =pi?
    cout << "Inicjacja:  " << elapsed_time(t0, t1) << " ms" << endl;
    cout << "Generacja:  " << elapsed_time(t1, t2) << " ms" << endl;
    cout << "Integracja: " << elapsed_time(t2, t3) << " ms" << endl;
    cout << "R A Z E M : " << elapsed_time(t0, t3) << " ms" << endl;
#ifdef _WIN32
    if (IsDebuggerPresent()) getchar();
#endif
    return 0;
}